#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

#define MAX_RESULTS 150000000 // Maximum number of triangles to store

struct Triangle {
    long long a, b, c;
};

// Efficient integer square root function for GPU
__device__ unsigned long long isqrt(unsigned long long x) {
    unsigned long long res = 0;
    unsigned long long bit = 1ULL << 62;

    while (bit > x) bit >>= 2;

    while (bit != 0) {
        if (x >= res + bit) {
            x -= res + bit;
            res = (res >> 1) + bit;
        } else {
            res >>= 1;
        }
        bit >>= 2;
    }
    return res;
}

// Kernel for finding triangles
__global__ void findTrianglesKernel(int k, long long MAX_PERIMETER, Triangle* d_results, unsigned long long* d_count) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long totalThreads = gridDim.x * blockDim.x;

    long long b_min = 1;
    long long b_max = MAX_PERIMETER / 2;

    for (long long b = b_min + idx; b <= b_max; b += totalThreads) {
        for (long long c = b; c <= MAX_PERIMETER - b - 1; ++c) {
            unsigned long long D = (unsigned long long)(b - c) * (b - c) + 4 * k * b * c;
            unsigned long long s = isqrt(D);
            if (s * s != D) continue;

            long long a_numerator = - (b + c) + s;
            if (a_numerator <= 0 || a_numerator % 2 != 0) continue;
            long long a = a_numerator / 2;
            if (a > b) continue;

            if (a + b <= c || a + c <= b || b + c <= a) continue;
            if (a + b + c > MAX_PERIMETER) continue;

            unsigned long long ratio_numerator = (a + b) * (a + c);
            unsigned long long ratio_denominator = b * c;
            if (ratio_numerator != (unsigned long long)k * ratio_denominator) continue;

            unsigned long long pos = atomicAdd(d_count, 1ULL);
            if (pos < MAX_RESULTS) {
                d_results[pos].a = a;
                d_results[pos].b = b;
                d_results[pos].c = c;
            }
        }
    }
}

// Equilateral triangles kernel
__global__ void findEquilateralTrianglesKernel(long long MAX_PERIMETER, Triangle* d_results, unsigned long long* d_count) {
    unsigned long long idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long totalThreads = gridDim.x * blockDim.x;

    long long max_a = MAX_PERIMETER / 3;

    for (long long a = idx + 1; a <= max_a; a += totalThreads) {
        unsigned long long pos = atomicAdd(d_count, 1ULL);
        if (pos < MAX_RESULTS) {
            d_results[pos].a = a;
            d_results[pos].b = a;
            d_results[pos].c = a;
        }
    }
}

int main() {
    // User input for maximum perimeter
    long long MAX_PERIMETER;
    std::cout << "Enter the maximum perimeter: ";
    std::cin >> MAX_PERIMETER;

    auto start_time = std::chrono::high_resolution_clock::now();

    // CUDA settings
    int device = 0;
    hipError_t err = hipSetDevice(device);
    if (err != hipSuccess) {
        std::cerr << "Error setting CUDA device: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Compute optimal block size and grid size for findTrianglesKernel
    int blockSizeTriangles;
    int minGridSizeTriangles;
    size_t dynamicSMemSize = 0;

    err = hipOccupancyMaxPotentialBlockSize(
        &minGridSizeTriangles,
        &blockSizeTriangles,
        findTrianglesKernel,
        dynamicSMemSize,
        0);
    if (err != hipSuccess) {
        std::cerr << "Error in hipOccupancyMaxPotentialBlockSize for findTrianglesKernel: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Calculate total combinations
    long long b_min = 1;
    long long b_max = MAX_PERIMETER / 2;
    unsigned long long total_b = b_max - b_min + 1;

    // For the loops in the kernel
    unsigned long long totalThreadsNeeded = total_b;
    int gridSizeTriangles = (totalThreadsNeeded + blockSizeTriangles - 1) / blockSizeTriangles;

    // Limit grid size to maximum allowed
    int deviceMaxGridSizeX;
    err = hipDeviceGetAttribute(&deviceMaxGridSizeX, hipDeviceAttributeMaxGridDimX, device);
    if (err != hipSuccess) {
        std::cerr << "Error getting device attribute: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    if (gridSizeTriangles > deviceMaxGridSizeX) {
        gridSizeTriangles = deviceMaxGridSizeX;
    }

    // Print the computed block size and grid size for findTrianglesKernel
    std::cout << "Optimal block size for findTrianglesKernel: " << blockSizeTriangles << std::endl;
    std::cout << "Calculated grid size for findTrianglesKernel: " << gridSizeTriangles << std::endl;

    // Similarly, compute for findEquilateralTrianglesKernel
    int blockSizeEquilateral;
    int minGridSizeEquilateral;

    err = hipOccupancyMaxPotentialBlockSize(
        &minGridSizeEquilateral,
        &blockSizeEquilateral,
        findEquilateralTrianglesKernel,
        dynamicSMemSize,
        0);
    if (err != hipSuccess) {
        std::cerr << "Error in hipOccupancyMaxPotentialBlockSize for findEquilateralTrianglesKernel: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    long long max_a = MAX_PERIMETER / 3;
    unsigned long long totalThreadsNeededEquilateral = max_a;
    int gridSizeEquilateral = (totalThreadsNeededEquilateral + blockSizeEquilateral - 1) / blockSizeEquilateral;

    if (gridSizeEquilateral > deviceMaxGridSizeX) {
        gridSizeEquilateral = deviceMaxGridSizeX;
    }

    // Print the computed block size and grid size for findEquilateralTrianglesKernel
    std::cout << "Optimal block size for findEquilateralTrianglesKernel: " << blockSizeEquilateral << std::endl;
    std::cout << "Calculated grid size for findEquilateralTrianglesKernel: " << gridSizeEquilateral << std::endl;

    // Allocate memory for results
    Triangle* d_results;
    err = hipMalloc((void**)&d_results, MAX_RESULTS * sizeof(Triangle));
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory for results: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    unsigned long long* d_count;
    err = hipMalloc((void**)&d_count, sizeof(unsigned long long));
    if (err != hipSuccess) {
        std::cerr << "Error allocating device memory for count: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        return -1;
    }

    err = hipMemset(d_count, 0, sizeof(unsigned long long));
    if (err != hipSuccess) {
        std::cerr << "Error initializing device memory for count: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    // Launch kernels with optimized parameters
    findTrianglesKernel<<<gridSizeTriangles, blockSizeTriangles>>>(2, MAX_PERIMETER, d_results, d_count);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error for k=2: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    findTrianglesKernel<<<gridSizeTriangles, blockSizeTriangles>>>(3, MAX_PERIMETER, d_results, d_count);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error for k=3: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    findEquilateralTrianglesKernel<<<gridSizeEquilateral, blockSizeEquilateral>>>(MAX_PERIMETER, d_results, d_count);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch error for equilateral triangles: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error during device synchronization: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    // Copy result count back to host
    unsigned long long h_count = 0;
    err = hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying count from device to host: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    if (h_count > MAX_RESULTS) h_count = MAX_RESULTS;

    // Copy results back to host
    std::vector<Triangle> validTriangles(h_count);
    err = hipMemcpy(validTriangles.data(), d_results, h_count * sizeof(Triangle), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Error copying results from device to host: " << hipGetErrorString(err) << std::endl;
        hipFree(d_results);
        hipFree(d_count);
        return -1;
    }

    // Free device memory
    hipFree(d_results);
    hipFree(d_count);

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end_time - start_time;

    // Output results
    std::cout << "Found " << h_count << " valid triangles." << std::endl;
    std::cout << "Time taken: " << elapsed.count() << " seconds." << std::endl;

    // Optionally, print triangles
    /*
    for (const auto& triangle : validTriangles) {
        std::cout << "a = " << triangle.a << ", b = " << triangle.b << ", c = " << triangle.c << std::endl;
    }
    */

    return 0;
}